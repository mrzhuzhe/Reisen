#include "hip/hip_runtime.h"
#include "pba2D.h"

#define BLOCKSIZE 32

short2 **pbaTextures;
short2 *margin;

size_t pbaMemSize;
int pbaTexSize;

#include "pba2DKernel.h"

#define ULL unsigned long long 
void pba2DInitialization(int textureSize, int phase1Band)
{
    pbaTexSize = textureSize;
    pbaMemSize = (ULL)pbaTexSize * (ULL)pbaTexSize * (ULL)sizeof(short2);
    pbaTextures = (short2 **)malloc(2*sizeof(short2 *));

    hipMalloc((void **)&pbaTextures[0], pbaMemSize);
    hipMalloc((void **)&pbaTextures[1], pbaMemSize);
    hipMalloc((void **) &margin, 2ULL*(ULL)phase1Band*(ULL)pbaTexSize*sizeof(short2));
}
#undef ULL

// Deallocate all allocated memory
void pba2DDeinitialization()
{
    hipFree(pbaTextures[0]); 
    hipFree(pbaTextures[1]); 
    hipFree(margin);

    free(pbaTextures); 
}

// Copy input to GPU 
void pba2DInitializeInput(short *input)
{
    hipMemcpy(pbaTextures[0], input, pbaMemSize, hipMemcpyHostToDevice); 
}


// Phase 1 of PBA. m1 must divides texture size and equal or less than size / 64
void pba2DPhase1(int m1) 
{
    dim3 block = dim3(BLOCKSIZE);   
    dim3 grid = dim3(pbaTexSize / block.x, m1); 

    kernelFloodDown<<< grid, block >>>(pbaTextures[0], pbaTextures[0], pbaTexSize, pbaTexSize / m1); 

    kernelFloodUp<<< grid, block >>>(pbaTextures[0], pbaTextures[0], pbaTexSize, pbaTexSize / m1); 

    kernelPropagateInterband<<< grid, block >>>(pbaTextures[0], margin, pbaTexSize, pbaTexSize / m1);

    kernelUpdateVertical<<< grid, block >>>(pbaTextures[0], margin, pbaTextures[1], pbaTexSize, pbaTexSize / m1);
}

// Phase 2 of PBA. m2 must divides texture size
void pba2DPhase2(int m2) 
{
    // Compute proximate points locally in each band
    dim3 block = dim3(BLOCKSIZE);
    dim3 grid = dim3(pbaTexSize / block.x, m2);

    kernelProximatePoints<<< grid, block >>>(pbaTextures[1], pbaTextures[0], pbaTexSize, pbaTexSize / m2); 

    kernelCreateForwardPointers<<< grid, block >>>(pbaTextures[0], pbaTextures[0], pbaTexSize, pbaTexSize / m2); 

    // Repeatly merging two bands into one
    for (int noBand = m2; noBand > 1; noBand /= 2) {
        grid = dim3(pbaTexSize / block.x, noBand / 2); 
        kernelMergeBands<<< grid, block >>>(pbaTextures[1], pbaTextures[0], pbaTextures[0], pbaTexSize, pbaTexSize / noBand); 
    }

    // Replace the forward link with the X coordinate of the seed to remove
    // the need of looking at the other texture. We need it for coloring.
    grid = dim3(pbaTexSize / block.x, pbaTexSize); 
    kernelDoubleToSingleList<<< grid, block >>>(pbaTextures[1], pbaTextures[0], pbaTextures[0], pbaTexSize); 
}

// Phase 3 of PBA. m3 must divides texture size and equal or less than 64
void pba2DPhase3(int m3) 
{
    dim3 block = dim3(BLOCKSIZE, m3); 
    dim3 grid = dim3(pbaTexSize / block.x);
    
    kernelColor<<< grid, block >>>(pbaTextures[0], pbaTextures[1], pbaTexSize); 
}

void pba2DCompute(int m1, int m2, int m3)
{
    pba2DPhase1(m1);  

    pba2DPhase2(m2); 

    pba2DPhase3(m3); 
}

// Compute 2D Voronoi diagram
// Input: a 2D texture. Each pixel is represented as two "short" integer. 
//    For each site at (x, y), the pixel at coordinate (x, y) should contain 
//    the pair (x, y). Pixels that are not sites should contain the pair (MARKER, MARKER)
// See original paper for the effect of the three parameters: m1, m2, m3
// Parameters must divide textureSize
void pba2DVoronoiDiagram(short *input, short *output, int m1, int m2, int m3) 
{
    // Initialization
    pba2DInitializeInput(input); 

    // Computation
    pba2DCompute(m1, m2, m3); 

    // Copy back the result
    hipMemcpy(output, pbaTextures[1], pbaMemSize, hipMemcpyDeviceToHost); 
}